#include <unistd.h>
#include <limits.h>

#include <iostream>
#include <iomanip>
#include <random>
#include <array>

#include <mpi.h>
#include <hip/hip_runtime.h>

void find_cuda_devices(int rank) {

    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess) {
        std::cerr << "Error in hipGetDeviceCount: " << hipGetErrorString(error) << std::endl;
        exit(1);
    }

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        error = hipGetDeviceProperties(&deviceProp, i);

        if (error != hipSuccess) {
            std::cerr << "Error in hipGetDeviceProperties: " << hipGetErrorString(error) << std::endl;
            exit(1);
        }

        std::cout << "rank=" << rank << " is using device=" << i << " name=" << deviceProp.name << " uuid=";

        for (int j = 0; j < 16; ++j) {
            std::cout << std::hex << std::setw(2) << std::setfill('0') << (deviceProp.uuid.bytes[j] & 0xFF);
        }

        std::cout << std::dec << std::endl;
    }

}

int main(int argc, char *argv[]) {
    char hostname[HOST_NAME_MAX];
    gethostname(hostname, HOST_NAME_MAX);

    MPI_Init(&argc, &argv);

    int rank, size;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank );
    MPI_Comm_size( MPI_COMM_WORLD, &size );

    std::cout << hostname << " is running rank=" << rank << "/" << size << std::endl;

    find_cuda_devices(rank);

    // generate a 'largeish' array of random numbers to broadcast to simulate real data.
    const size_t array_size = 2UL * 1024 * 1024 / sizeof(double);
    std::array<double, array_size> data;

    if (rank == 0) {
        std::uniform_real_distribution<double> distribution(0.0, 1.0);
        std::default_random_engine random_engine;

        for (size_t i = 0; i < array_size; ++i)
            data[i] = distribution(random_engine);
    }

    MPI_Bcast(data.data(), array_size, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    std::cout << rank << " received data" << std::endl;

    MPI_Finalize();
    return 0;
}